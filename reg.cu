#include "hip/hip_runtime.h"
/**
 * @file reg.cu
 * @brief Contains the code for the CUDA version of the REG algorithm
 */

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
// #include "utils.cuh"

#define SEQ "GTTAAGTTAAGT" //"GTTAAGTTAA"
#define SEQ_LEN strlen(SEQ)

__host__ __device__ int ATCG_to_int(char c)
{
    switch (c)
    {
    case 65: //'A'
        return 0;
        break;
    case 84: //'T'
        return 1;
        break;
    case 67: //'C'
        return 2;
        break;
    case 71: //'G':
        return 3;
        break;
    default:
        return c;
        break;
    }
}

/*
 * @brief Open the sequence file.
 * @param *path path to the file
 * @return FILE* file pointer
 */
FILE *openSequence(const char *path)
{
    FILE *fp;
    fp = fopen(path, "r");
    if (fp == NULL)
    {
        printf("Error while opening file : %s\n", path);
        exit(EXIT_FAILURE);
    }
    return fp;
}

/*
 * @brief Get the size of the file.
 * @param *fp file pointer
 * @return long size of the file
 */
long get_size_file(FILE *fp)
{
    fseek(fp, 0L, SEEK_END);
    long size = ftell(fp);
    rewind(fp);
    return size;
}

/*
 * @brief Return the int corresponding to the given ADN sequence.
 * @param *seq sequence to convert
 * @return int converted sequence
 */
int code_seq_bin(const char *seq)
{
    int bin = ATCG_to_int(seq[0]);
    for (int i = 1; i < strlen(seq); i++)
    {
        bin = bin << 2;
        bin = bin | ATCG_to_int(seq[i]);
    }
    return bin;
}

/*
 * @brief Return the time difference between two timeval.
 * @param *start start timeval
 * @param *end end timeval
 * @return float time difference
 */
float time_diff(struct timeval *start, struct timeval *end)
{
    return (end->tv_sec - start->tv_sec) + 1e-6 * (end->tv_usec - start->tv_usec);
}

__global__ void researchThread(char *part, long size, int seq_hash, int effaceur)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int nb_threads = blockDim.x * blockDim.y * gridDim.x * gridDim.y;
    long part_size = size / nb_threads;
    long start = id == 0 ? id * part_size : id * part_size - SEQ_LEN + 1;
    long end = id * part_size + part_size;

    // Initialize the first window
    long i = start;
    int win = ATCG_to_int(part[i]);
    for (short int j = 1; j < SEQ_LEN; j++)
    {
        i++;
        win = win << 2;
        win = win | ATCG_to_int(part[i]);
    }
    if (win == seq_hash)
    {
        printf("Thread %d found at position %ld\n", id, i);
    }

    // Slide the window and compare
    while (i < end)
    {
        i++;
        int c = part[i];
        while (c == 78) // 'N'
        {
            i++;
            c = part[i];
        }
        if (c == 10) //'\n'
        {
            i++;
            c = part[i];
        }

        win = win << 2;
        win = win | ATCG_to_int(c);
        win = win & effaceur;

        if (win == seq_hash)
        {
            printf("Thread %d found at position %ld\n", id, i);
        }
    }
}

int main()
{
    // Initialization
    printf("Researching '%s' :\n", SEQ);
    struct timeval start_loading, end_loading, start_loading_gpu, end_loading_gpu, start_searching, end_searching;

    int seq_hash = code_seq_bin(SEQ);
    int effaceur = (int)(pow(2, 2 * SEQ_LEN) - 1);

    // Get the file and its size
    gettimeofday(&start_loading, NULL);
    const char *path = "sequences/GRCH38";
    FILE *file = openSequence(path);
    long size = get_size_file(file);

    // Load the file in memory
    char *buffer = (char *)malloc(size);
    size_t bytesRead = fread(buffer, 1, size, file);
    fclose(file);
    gettimeofday(&end_loading, NULL);
    printf("Loaded %ld octets in %fs\n", size, time_diff(&start_loading, &end_loading));

    // Copy the buffer to the GPU
    gettimeofday(&start_loading_gpu, NULL);
    char *d_buffer; //= (char *)malloc(size);
    hipMalloc((void **)&d_buffer, size);
    hipMemcpy(d_buffer, buffer, size, hipMemcpyHostToDevice);
    gettimeofday(&end_loading_gpu, NULL);
    printf("Loaded %ld octets in %fs on the GPU\n", size, time_diff(&start_loading_gpu, &end_loading_gpu));

    // Config threads
    int threads_per_blocks = 128; // Adjust to the GPU
    int blocks_per_grid = 10;     // Adjust to the GPU

    // launch threads
    gettimeofday(&start_searching, NULL);
    researchThread<<<blocks_per_grid, threads_per_blocks>>>(d_buffer, size, seq_hash, effaceur);

    // Wait for threads to finish
    hipDeviceSynchronize();
    gettimeofday(&end_searching, NULL);

    // printf("Found %d times\n", found);
    printf("Time taken: %f seconds\n", time_diff(&start_searching, &end_searching));

    free(buffer);
    hipFree(d_buffer);
    return EXIT_SUCCESS;
}
